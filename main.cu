#include "hip/hip_runtime.h"
// c, c++
#include <cstdio>
#include <cstdint>
#include <cstddef>
#include <sstream>
#include <chrono>
#include <iostream>
#include <vector>
// cuda, cudnn
#include <hipDNN.h>
#include <hipblas.h>

#define checkCUDNN(expression)                                         \
{                                                                      \
    hipdnnStatus_t status = (expression);                               \
    if (status != HIPDNN_STATUS_SUCCESS) {                              \
      std::cerr << "Error on line " << __LINE__ << ": "                \
      << hipdnnGetErrorString(status) << std::endl;                     \
	  std::exit(EXIT_FAILURE);                                         \
    }                                                                  \
}

#define checkCudaErrors(expression)                                    \
{                                                                      \
    uint32_t status = (expression);                                    \
    if (status != 0) {                                                 \
      std::cerr << "Error on line " << __LINE__ << ": "                \
      << "Cuda failure: " << status << std::endl;                      \
	  std::exit(EXIT_FAILURE);                                         \
    }                                                                  \
}

#define bswap(x) __builtin_bswap32(x);

struct ImageMetaData
{
	uint32_t magicNum;
	uint32_t size;
	uint32_t height;
	uint32_t width;

	void swap()
	{
		magicNum = bswap(magicNum);
		size = bswap(size);
		height = bswap(height);
		width = bswap(width);
	}
};

struct LabelMetaData
{
	uint32_t magicNum;
	uint32_t size;

	void swap()
	{
		magicNum = bswap(magicNum);
		size = bswap(size);
	}
};

struct ConvolutionLayer
{
	int channelsIn, channelsOut, kernelSize;
	int widthIn, heightIn, widthOut, heightOut;

	std::vector<float> weight, bias;

	ConvolutionLayer(int channelsIn_, int channelsOut_, int kernelSize_, int width, int height) :
		             weight(channelsIn_ * kernelSize_ * kernelSize_ * channelsOut_),
		             bias(channelsOut_)
	{
		channelsIn = channelsIn_;
		channelsOut = channelsOut_;
		kernelSize = kernelSize_;
		widthIn = width;
		heightIn = height;
		widthOut = width - kernelSize_ + 1;
		heightOut = height - kernelSize_ + 1;
	}

	bool loadWeights(const char *filePrefix)
	{
		std::stringstream s1, s2;
		s1 << filePrefix << ".bin";
		s2 << filePrefix << ".bias.bin";

		FILE *fp = fopen(s1.str().c_str(), "rb");
		if (!fp) return false;
		fread(&weight[0], sizeof(float), channelsIn * channelsOut * kernelSize * kernelSize, fp);
		fclose(fp);

		fp = fopen(s2.str().c_str(), "rb");
		if (!fp) return false;
		fread(&bias[0], sizeof(float), channelsOut, fp);
		fclose(fp);

		return true;
	}
};

struct PoolingLayer
{
	int size, stride;
	PoolingLayer(int size_, int stride_) : size(size_), stride(stride_) {}
};

struct FullyConnectedLayer
{
	int inputs, outputs;
	std::vector<float> weight, bias;

	FullyConnectedLayer(int inputs_, int outputs_) :
		                inputs(inputs_), outputs(outputs_),
		                weight(inputs_ * outputs_), bias(outputs_) {}

	bool loadWeights(const char *filePrefix)
	{
		std::stringstream s1, s2;
		s1 << filePrefix << ".bin";
		s2 << filePrefix << ".bias.bin";

		FILE *fp = fopen(s1.str().c_str(), "rb");
		if (!fp) return false;
		fread(&weight[0], sizeof(float), inputs * outputs, fp);
		fclose(fp);

		fp = fopen(s2.str().c_str(), "rb");
		if (!fp) return false;
		fread(&bias[0], sizeof(float), outputs, fp);
		fclose(fp);

		return true;
	}
};

struct Lenet
{
	hipdnnHandle_t cudnnHandle;
	hipblasHandle_t cublasHandle;

	hipdnnTensorDescriptor_t inputTensor;

	hipdnnTensorDescriptor_t conv1_OutputTensor, conv2_OutputTensor, conv1_BiasTensor, conv2_BiasTensor;
	hipdnnFilterDescriptor_t conv1_FilterDesc, conv2_FilterDesc;
	hipdnnConvolutionDescriptor_t conv1_Desc, conv2_Desc;
	hipdnnActivationDescriptor_t conv1_ActDesc, conv2_ActDesc;

	hipdnnTensorDescriptor_t pool1_OutputTensor, pool2_OutputTensor;
	hipdnnPoolingDescriptor_t pool1_Desc, pool2_Desc;

	hipdnnTensorDescriptor_t fc1_OutputTensor, fc2_OutputTensor;
	hipdnnActivationDescriptor_t fc1_ActDesc;

	hipdnnConvolutionFwdAlgo_t conv1_AlgoDesc, conv2_AlgoDesc;

	size_t m_workSpaceSize;

	ConvolutionLayer *m_conv1, *m_conv2;
	PoolingLayer *m_pool1, *m_pool2;
	FullyConnectedLayer *m_fc1, *m_fc2;

	float *conv1_data, *conv1_relu_data, *pool1_data,
		  *conv2_data, *conv2_relu_data, *pool2_data,
		  *fc1_data, *fc1relu_data, *fc2_data;

	float *conv1_weight, *conv1_bias, *conv2_weight, *conv2_bias;
	float *fc1_weight, *fc1_bias, *fc2_weight, *fc2_bias;

	float *vecter;
	void *workspace;

	Lenet(int channels, int width, int height)
	{
		m_conv1 = new ConvolutionLayer((int)channels, 6, 5, (int)width, (int)height);
		m_pool1 = new PoolingLayer(2, 2);
		m_conv2 = new ConvolutionLayer(m_conv1->channelsOut, 16, 5, m_conv1->widthOut / m_pool1->stride, m_conv1->heightOut / m_pool1->stride);
		m_pool2 = new PoolingLayer(2, 2);
		m_fc1 = new FullyConnectedLayer((m_conv2->channelsOut*m_conv2->widthOut*m_conv2->heightOut) / (m_pool2->stride * m_pool2->stride), 500);
		m_fc2 = new FullyConnectedLayer(m_fc1->outputs, 10);
		m_conv1->loadWeights("conv1");
		m_conv2->loadWeights("conv2");
		m_fc1->loadWeights("fc1");
		m_fc2->loadWeights("fc2");

		m_workSpaceSize = 0;
		Setup();
		MemoryLocate();
	}

	~Lenet()
	{
		checkCudaErrors(hipblasDestroy(cublasHandle));
		checkCUDNN(hipdnnDestroy(cudnnHandle));
		checkCUDNN(hipdnnDestroyTensorDescriptor(inputTensor));
		checkCUDNN(hipdnnDestroyTensorDescriptor(conv1_OutputTensor));
		checkCUDNN(hipdnnDestroyTensorDescriptor(conv2_OutputTensor));
		checkCUDNN(hipdnnDestroyTensorDescriptor(conv1_BiasTensor));
		checkCUDNN(hipdnnDestroyTensorDescriptor(conv2_BiasTensor));
		checkCUDNN(hipdnnDestroyFilterDescriptor(conv1_FilterDesc));
		checkCUDNN(hipdnnDestroyFilterDescriptor(conv2_FilterDesc));
		checkCUDNN(hipdnnDestroyConvolutionDescriptor(conv1_Desc));
		checkCUDNN(hipdnnDestroyConvolutionDescriptor(conv2_Desc));
		checkCUDNN(hipdnnDestroyActivationDescriptor(conv1_ActDesc));
		checkCUDNN(hipdnnDestroyActivationDescriptor(conv2_ActDesc));
		checkCUDNN(hipdnnDestroyTensorDescriptor(pool1_OutputTensor));
		checkCUDNN(hipdnnDestroyTensorDescriptor(pool2_OutputTensor));
		checkCUDNN(hipdnnDestroyPoolingDescriptor(pool1_Desc));
		checkCUDNN(hipdnnDestroyPoolingDescriptor(pool2_Desc));
		checkCUDNN(hipdnnDestroyTensorDescriptor(fc1_OutputTensor));
		checkCUDNN(hipdnnDestroyTensorDescriptor(fc2_OutputTensor));
		checkCUDNN(hipdnnDestroyActivationDescriptor(fc1_ActDesc));

		checkCudaErrors(hipFree(conv1_data));
		checkCudaErrors(hipFree(conv1_relu_data));
		checkCudaErrors(hipFree(pool1_data));
		checkCudaErrors(hipFree(conv2_data));
		checkCudaErrors(hipFree(conv2_relu_data));
		checkCudaErrors(hipFree(pool2_data));
		checkCudaErrors(hipFree(fc1_data));
		checkCudaErrors(hipFree(fc1relu_data));
		checkCudaErrors(hipFree(fc2_data));

		checkCudaErrors(hipFree(conv1_weight));
		checkCudaErrors(hipFree(conv1_bias));
		checkCudaErrors(hipFree(conv2_weight));
		checkCudaErrors(hipFree(conv2_bias));
		checkCudaErrors(hipFree(fc1_weight));
		checkCudaErrors(hipFree(fc1_bias));
		checkCudaErrors(hipFree(fc2_weight));
		checkCudaErrors(hipFree(fc2_bias));

		checkCudaErrors(hipFree(vecter));
		checkCudaErrors(hipFree(workspace));

		delete m_conv1, m_conv2, m_pool1, m_pool2, m_fc1, m_fc2;
	}

	void Setup()
	{
		int batchSize = 1;
		size_t sizeBytes = 0;

		checkCudaErrors(hipblasCreate(&cublasHandle));
		checkCUDNN(hipdnnCreate(&cudnnHandle));
                     
		// conv 1
		checkCUDNN(hipdnnCreateTensorDescriptor(&conv1_BiasTensor));
		checkCUDNN(hipdnnCreateActivationDescriptor(&conv1_ActDesc));
		checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensor));
		checkCUDNN(hipdnnCreateFilterDescriptor(&conv1_FilterDesc));
		checkCUDNN(hipdnnCreateConvolutionDescriptor(&conv1_Desc));
		checkCUDNN(hipdnnCreateTensorDescriptor(&conv1_OutputTensor));

		int conv1_channelsIn = m_conv1->channelsIn;
		int conv1_channelsOut = m_conv1->channelsOut;
		int conv1_heightIn = m_conv1->heightIn;
		int conv1_widthIn = m_conv1->widthIn;
		int conv1_kernelSize = m_conv1->kernelSize;

		checkCUDNN(hipdnnSetTensor4dDescriptor(conv1_BiasTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
			                                  1, conv1_channelsOut,
			                                  1, 1));

		checkCUDNN(hipdnnSetActivationDescriptor(conv1_ActDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.01));

		checkCUDNN(hipdnnSetTensor4dDescriptor(inputTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
			                                  1, conv1_channelsIn,
			                                  conv1_heightIn, conv1_widthIn));

		checkCUDNN(hipdnnSetFilter4dDescriptor(conv1_FilterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
			                                  conv1_channelsOut, conv1_channelsIn,
			                                  conv1_kernelSize, conv1_kernelSize));

		checkCUDNN(hipdnnSetConvolution2dDescriptor(conv1_Desc, 0, 0, 1, 1, 1, 1,
			                                       HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

		checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(conv1_Desc, inputTensor, conv1_FilterDesc,
			                                             &batchSize, &conv1_channelsIn,
			                                             &conv1_heightIn, &conv1_widthIn));

		checkCUDNN(hipdnnSetTensor4dDescriptor(conv1_OutputTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
			                                  1, conv1_channelsIn,
			                                  conv1_heightIn, conv1_widthIn));

		checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnnHandle,
			                                           inputTensor,
			                                           conv1_FilterDesc,
			                                           conv1_Desc,
			                                           conv1_OutputTensor,
			                                           HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
			                                           0,
			                                           &conv1_AlgoDesc));

		checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle,
			                                               inputTensor,
			                                               conv1_FilterDesc,
		                                                   conv1_Desc,
			                                               conv1_OutputTensor,
			                                               conv1_AlgoDesc,
			                                               &sizeBytes));

		m_workSpaceSize = std::max(m_workSpaceSize, sizeBytes);

		// pool 1
		checkCUDNN(hipdnnCreatePoolingDescriptor(&pool1_Desc));

		checkCUDNN(hipdnnSetPooling2dDescriptor(pool1_Desc, HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN,
			                                   m_pool1->size, m_pool1->size,
			                                   0, 0,
			                                   m_pool1->stride, m_pool1->stride));

		// conv 2
		checkCUDNN(hipdnnCreateTensorDescriptor(&conv2_BiasTensor));
		checkCUDNN(hipdnnCreateActivationDescriptor(&conv2_ActDesc));
		checkCUDNN(hipdnnCreateTensorDescriptor(&pool1_OutputTensor));
		checkCUDNN(hipdnnCreateFilterDescriptor(&conv2_FilterDesc));
		checkCUDNN(hipdnnCreateConvolutionDescriptor(&conv2_Desc));
		checkCUDNN(hipdnnCreateTensorDescriptor(&conv2_OutputTensor));

		int conv2_channelsIn = m_conv2->channelsIn;
		int conv2_channelsOut = m_conv2->channelsOut;
		int conv2_heightIn = m_conv2->heightIn;
		int conv2_widthIn = m_conv2->widthIn;
		int conv2_kernelSize = m_conv2->kernelSize;

		checkCUDNN(hipdnnSetTensor4dDescriptor(conv2_BiasTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
			                                  1, m_conv2->channelsOut,
			                                  1, 1));

		checkCUDNN(hipdnnSetActivationDescriptor(conv2_ActDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.01));

		checkCUDNN(hipdnnSetTensor4dDescriptor(pool1_OutputTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
			                                  1, conv2_channelsIn,
			                                  conv2_heightIn, conv2_widthIn));

		checkCUDNN(hipdnnSetFilter4dDescriptor(conv2_FilterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
			                                  conv2_channelsOut, conv2_channelsIn,
			                                  conv2_kernelSize, conv2_kernelSize));

		checkCUDNN(hipdnnSetConvolution2dDescriptor(conv2_Desc, 0, 0, 1, 1, 1, 1,
			                                       HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

		checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(conv2_Desc, pool1_OutputTensor, conv2_FilterDesc,
			                                             &batchSize, &conv2_channelsIn,
			                                             &conv2_heightIn, &conv2_widthIn));

		checkCUDNN(hipdnnSetTensor4dDescriptor(conv2_OutputTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
			                                  1, conv2_channelsIn,
			                                  conv2_heightIn, conv2_widthIn));

		checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnnHandle,
			                                           pool1_OutputTensor,
			                                           conv2_FilterDesc,
			                                           conv2_Desc,
			                                           conv2_OutputTensor,
			                                           HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
			                                           0,
			                                           &conv2_AlgoDesc));

		checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle,
			                                               pool1_OutputTensor,
			                                               conv2_FilterDesc,
		                                                   conv2_Desc,
			                                               conv2_OutputTensor,
			                                               conv2_AlgoDesc,
			                                               &sizeBytes));

		m_workSpaceSize = std::max(m_workSpaceSize, sizeBytes);

		// pool 2
		checkCUDNN(hipdnnCreateTensorDescriptor(&pool2_OutputTensor));
		checkCUDNN(hipdnnCreatePoolingDescriptor(&pool2_Desc));

		checkCUDNN(hipdnnSetTensor4dDescriptor(pool2_OutputTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
			                                  1, m_conv2->channelsOut,
			                                  m_conv2->heightOut / m_pool2->stride,
			                                  m_conv2->widthOut / m_pool2->stride));

		checkCUDNN(hipdnnSetPooling2dDescriptor(pool2_Desc, HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN,
			                                   m_pool2->size, m_pool2->size,
			                                   0, 0,
			                                   m_pool2->stride, m_pool2->stride));
		// fc 1
		checkCUDNN(hipdnnCreateTensorDescriptor(&fc1_OutputTensor));
		checkCUDNN(hipdnnCreateActivationDescriptor(&fc1_ActDesc));


		checkCUDNN(hipdnnSetTensor4dDescriptor(fc1_OutputTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
			                                  1, m_fc1->outputs, 1, 1));

		checkCUDNN(hipdnnSetActivationDescriptor(fc1_ActDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.01));

		// fc 2
		checkCUDNN(hipdnnCreateTensorDescriptor(&fc2_OutputTensor));
		
		checkCUDNN(hipdnnSetTensor4dDescriptor(fc2_OutputTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
			                                  1, m_fc2->outputs, 1, 1));
	}

	void MemoryLocate()
	{
		checkCudaErrors(hipMalloc(&conv1_data, sizeof(float)* m_conv1->channelsOut * m_conv1->heightOut                  * m_conv1->widthOut));
		checkCudaErrors(hipMalloc(&conv1_relu_data, sizeof(float) * m_conv1->channelsOut * m_conv1->heightOut                  * m_conv1->widthOut));
		checkCudaErrors(hipMalloc(&pool1_data, sizeof(float) * m_conv1->channelsOut * (m_conv1->heightOut / m_pool1->stride) * (m_conv1->widthOut / m_pool1->stride)));
		checkCudaErrors(hipMalloc(&conv2_data, sizeof(float) * m_conv2->channelsOut * m_conv2->heightOut                  * m_conv2->widthOut));
		checkCudaErrors(hipMalloc(&conv2_relu_data, sizeof(float) * m_conv2->channelsOut * m_conv2->heightOut                  * m_conv2->widthOut));
		checkCudaErrors(hipMalloc(&pool2_data, sizeof(float) * m_conv2->channelsOut * (m_conv2->heightOut / m_pool2->stride) * (m_conv2->widthOut / m_pool2->stride)));
		checkCudaErrors(hipMalloc(&fc1_data, sizeof(float) * m_fc1->outputs));
		checkCudaErrors(hipMalloc(&fc1relu_data, sizeof(float) * m_fc1->outputs));
		checkCudaErrors(hipMalloc(&fc2_data, sizeof(float) * m_fc2->outputs));

		checkCudaErrors(hipMalloc(&conv1_weight, sizeof(float) * m_conv1->weight.size()));
		checkCudaErrors(hipMalloc(&conv1_bias, sizeof(float) * m_conv1->bias.size()));
		checkCudaErrors(hipMalloc(&conv2_weight, sizeof(float) * m_conv2->weight.size()));
		checkCudaErrors(hipMalloc(&conv2_bias, sizeof(float) * m_conv2->bias.size()));
		checkCudaErrors(hipMalloc(&fc1_weight, sizeof(float) * m_fc1->weight.size()));
		checkCudaErrors(hipMalloc(&fc1_bias, sizeof(float) * m_fc1->bias.size()));
		checkCudaErrors(hipMalloc(&fc2_weight, sizeof(float) * m_fc2->weight.size()));
		checkCudaErrors(hipMalloc(&fc2_bias, sizeof(float) * m_fc2->bias.size()));

		checkCudaErrors(hipMalloc(&vecter, sizeof(float)));
		checkCudaErrors(hipMalloc(&workspace, m_workSpaceSize));

		checkCudaErrors(hipMemcpyAsync(conv1_weight, &m_conv1->weight[0], sizeof(float) * m_conv1->weight.size(), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpyAsync(conv1_bias, &m_conv1->bias[0], sizeof(float) * m_conv1->bias.size(), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpyAsync(conv2_weight, &m_conv2->weight[0], sizeof(float) * m_conv2->weight.size(), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpyAsync(conv2_bias, &m_conv2->bias[0], sizeof(float) * m_conv2->bias.size(), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpyAsync(fc1_weight, &m_fc1->weight[0], sizeof(float) * m_fc1->weight.size(), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpyAsync(fc1_bias, &m_fc1->bias[0], sizeof(float) * m_fc1->bias.size(), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpyAsync(fc2_weight, &m_fc2->weight[0], sizeof(float) * m_fc2->weight.size(), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpyAsync(fc2_bias, &m_fc2->bias[0], sizeof(float) * m_fc2->bias.size(), hipMemcpyHostToDevice));
	}

	void ForwardPass(float *input_data, float *softmax_data)
	{
		float alpha = 1.0f, beta = 0.0f;

		// conv 1
		checkCUDNN(hipdnnConvolutionForward(cudnnHandle, &alpha, inputTensor, input_data, 
			                               conv1_FilterDesc, conv1_weight, conv1_Desc, conv1_AlgoDesc,
			                               workspace, m_workSpaceSize, &beta, conv1_OutputTensor, conv1_data));

		checkCUDNN(hipdnnAddTensor(cudnnHandle, &alpha, conv1_BiasTensor, conv1_bias, &alpha, conv1_OutputTensor, conv1_data));

		checkCUDNN(hipdnnActivationForward(cudnnHandle, conv1_ActDesc, &alpha, 
			                              conv1_OutputTensor, conv1_data, &beta, conv1_OutputTensor, conv1_relu_data));

		// pool 1
		checkCUDNN(hipdnnPoolingForward(cudnnHandle, pool1_Desc, &alpha, conv1_OutputTensor, 
			                           conv1_relu_data, &beta, pool1_OutputTensor, pool1_data));

		// conv 2
		checkCUDNN(hipdnnConvolutionForward(cudnnHandle, &alpha, pool1_OutputTensor, pool1_data,
			                               conv2_FilterDesc, conv2_weight, conv2_Desc, conv2_AlgoDesc,
			                               workspace, m_workSpaceSize, &beta, conv2_OutputTensor, conv2_data));

		checkCUDNN(hipdnnAddTensor(cudnnHandle, &alpha, conv2_BiasTensor, conv1_bias, &alpha, conv2_OutputTensor, conv2_data));

		checkCUDNN(hipdnnActivationForward(cudnnHandle, conv2_ActDesc, &alpha,
			                              conv2_OutputTensor, conv2_data, &beta, conv2_OutputTensor, conv2_relu_data));

		// pool 2
		checkCUDNN(hipdnnPoolingForward(cudnnHandle, pool2_Desc, &alpha, conv2_OutputTensor,
			                           conv2_relu_data, &beta, pool2_OutputTensor, pool2_data));

		// fc 1
		checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
			                        m_fc1->outputs, 1, m_fc1->inputs,
			                        &alpha, fc1_weight, m_fc1->inputs,
			                        pool2_data, m_fc1->inputs, &beta,
			                        fc1_data, m_fc1->outputs));

		checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
			                        m_fc1->outputs, 1, 1,
			                        &alpha, fc1_bias, m_fc1->outputs,
			                        vecter, 1, &alpha,
			                        fc1_data, m_fc1->outputs));

		checkCUDNN(hipdnnActivationForward(cudnnHandle, fc1_ActDesc, &alpha,
			                              fc1_OutputTensor, fc1_data, &beta, fc1_OutputTensor, fc1relu_data));

		// fc 2
		checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
			                        m_fc2->outputs, 1, m_fc2->inputs,
			                        &alpha, fc2_weight, m_fc2->inputs,
			                        fc1relu_data, m_fc2->inputs, &beta,
			                        fc2_data, m_fc2->outputs));

		checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
			                        m_fc2->outputs, 1, 1,
			                        &alpha, fc2_bias, m_fc2->outputs,
			                        vecter, 1, &alpha,
			                        fc2_data, m_fc2->outputs));

		// softmax
		checkCUDNN(hipdnnSoftmaxForward(cudnnHandle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL,
			                           &alpha, fc2_OutputTensor, fc2_data, &beta, fc2_OutputTensor, softmax_data));
	}
};

int main(int argc, char **argv)
{
	checkCudaErrors(hipSetDevice(0));

	printf("Loading data...\n");

	FILE *fp_image = fopen("t10k-images-idx3-ubyte", "rb");
	if (!fp_image) return -1;

	ImageMetaData imageMetaData;
	if (fread(&imageMetaData, sizeof(ImageMetaData), 1, fp_image) != 1)
	{
		fclose(fp_image);
		return -1;
	}
	imageMetaData.swap();

	FILE *fp_label = fopen("t10k-labels-idx1-ubyte", "rb");
	if (!fp_label) return -1;

	LabelMetaData labelMetaData;
	if (fread(&labelMetaData, sizeof(LabelMetaData), 1, fp_label) != 1)
	{
		fclose(fp_label);
		return -1;
	}
	labelMetaData.swap();

	size_t width = imageMetaData.width;
	size_t height = imageMetaData.height;
	size_t testSize = imageMetaData.size;
	size_t channels = 1;

	std::vector<uint8_t> test_images(testSize * width * height * channels);
	if (fread(&test_images[0], sizeof(uint8_t), testSize * width * height, fp_image) != testSize * width * height)
	{
		fclose(fp_image);
		return -1;
	}

	std::vector<uint8_t> test_labels(testSize);
	if (fread(&test_labels[0], sizeof(uint8_t), testSize, fp_label) != testSize)
	{
		fclose(fp_label);
		return -1;
	}

	fclose(fp_image);
	fclose(fp_label);
	printf("Testing dataset size: %d\n", (int)testSize);

	Lenet LENET(channels, width, height);

	float *inputData, *softMax;
	checkCudaErrors(hipMalloc(&inputData, sizeof(float) * channels * height * width));
	checkCudaErrors(hipMalloc(&softMax, sizeof(float) * LENET.m_fc2->outputs));

	int correctCount = 0;
	auto t_start = std::chrono::high_resolution_clock::now();
	for (int i = 0; i < (int)testSize; ++i)
	{
		std::vector<float> image(width * height);

		for (int j = 0; j < width * height; ++j)
			image[j] = (float)test_images[i * width * height * channels + j] / 255.0f;

		checkCudaErrors(hipMemcpyAsync(inputData, &image[0], sizeof(float) * width * height, hipMemcpyHostToDevice));

		LENET.ForwardPass(inputData, softMax);

		std::vector<float> outVector(10);

		checkCudaErrors(hipMemcpy(&outVector[0], softMax, sizeof(float) * 10, hipMemcpyDeviceToHost));

		int predict = 0;
		for (int label = 1; label < 10; ++label)
		{
			if (outVector[predict] < outVector[label]) predict = label;
		}

		if (predict == test_labels[i])
			++correctCount;
	}
	auto t_end = std::chrono::high_resolution_clock::now();

	printf("Accuracy: %.2f %% \n", (float)correctCount / (float)testSize * 100.0f);
	printf("Cost time: %f ms\n", std::chrono::duration_cast<std::chrono::microseconds>(t_end - t_start).count() / 1000.0f);

	checkCudaErrors(hipFree(inputData));
	checkCudaErrors(hipFree(softMax));

	return 0;
}